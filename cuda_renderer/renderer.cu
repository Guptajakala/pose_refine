#include "hip/hip_runtime.h"
#include "renderer.h"
namespace cuda_renderer {

namespace normal_functor{  // similar to thrust
    __host__ __device__
    Model::float3 minus(const Model::float3& one, const Model::float3& the_other)
    {
        return {
            one.x - the_other.x,
            one.y - the_other.y,
            one.z - the_other.z
        };
    }
    __host__ __device__
    Model::float3 cross(const Model::float3& one, const Model::float3& the_other)
    {
        return {
            one.y*the_other.z - one.z*the_other.y,
            one.z*the_other.x - one.x*the_other.z,
            one.x*the_other.y - one.y*the_other.x
        };
    }
    __host__ __device__
    Model::float3 normalized(const Model::float3& one)
    {
        float norm = std::sqrt(one.x*one.x+one.y*one.y+one.z*one.z);
        return {
          one.x/norm,
          one.y/norm,
          one.z/norm
        };
    }

    __host__ __device__
    Model::float3 get_normal(const Model::Triangle& dev_tri)
    {
//      return normalized(cross(minus(dev_tri.v1, dev_tri.v0), minus(dev_tri.v1, dev_tri.v0)));

      // no need for normalizing?
      return (cross(minus(dev_tri.v1, dev_tri.v0), minus(dev_tri.v2, dev_tri.v0)));
    }

    __host__ __device__
    bool is_back(const Model::Triangle& dev_tri){
        return normal_functor::get_normal(dev_tri).z < 0;
    }
};

__host__ __device__
Model::float3 mat_mul_v(const Model::mat4x4& tran, const Model::float3& v){
    return {
        tran.a0*v.x + tran.a1*v.y + tran.a2*v.z + tran.a3,
        tran.b0*v.x + tran.b1*v.y + tran.b2*v.z + tran.b3,
        tran.c0*v.x + tran.c1*v.y + tran.c2*v.z + tran.c3,
    };
}

__host__ __device__
Model::Triangle transform_triangle(const Model::Triangle& dev_tri, const Model::mat4x4& tran){
    return {
        mat_mul_v(tran, (dev_tri.v0)),
        mat_mul_v(tran, (dev_tri.v1)),
        mat_mul_v(tran, (dev_tri.v2)),
    };
}

__host__ __device__
float calculateSignedArea(float* A, float* B, float* C){
    return 0.5f*((C[0]-A[0])*(B[1]-A[1]) - (B[0]-A[0])*(C[1]-A[1]));
}

__host__ __device__
Model::float3 barycentric(float* A, float* B, float* C, size_t* P) {

    float float_P[2] = {float(P[0]), float(P[1])};

    auto base_inv = 1/calculateSignedArea(A, B, C);
    auto beta = calculateSignedArea(A, float_P, C)*base_inv;
    auto gamma = calculateSignedArea(A, B, float_P)*base_inv;

    return {
        1.0f-beta-gamma,
        beta,
        gamma,
    };
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void print_cuda_memory_usage(){
    // show memory usage of GPU

    size_t free_byte ;
    size_t total_byte ;
    auto cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

    if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
    }

    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;
    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
        used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

struct max2zero_functor{

    max2zero_functor(){}

    __host__ __device__
    int32_t operator()(const int32_t& x) const
    {
      return (x==INT_MAX)? 0: x;
    }
};


__device__
void rasterization(const Model::Triangle dev_tri, Model::float3 last_row,
                                        int32_t* depth_entry, size_t width, size_t height, const Model::ROI roi){
    // refer to tiny renderer
    // https://github.com/ssloy/tinyrenderer/blob/master/our_gl.cpp
    float pts2[3][2];

    // viewport transform(0, 0, width, height)
    pts2[0][0] = dev_tri.v0.x/last_row.x*width/2.0f+width/2.0f; pts2[0][1] = dev_tri.v0.y/last_row.x*height/2.0f+height/2.0f;
    pts2[1][0] = dev_tri.v1.x/last_row.y*width/2.0f+width/2.0f; pts2[1][1] = dev_tri.v1.y/last_row.y*height/2.0f+height/2.0f;
    pts2[2][0] = dev_tri.v2.x/last_row.z*width/2.0f+width/2.0f; pts2[2][1] = dev_tri.v2.y/last_row.z*height/2.0f+height/2.0f;

    float bboxmin[2] = {FLT_MAX,  FLT_MAX};
    float bboxmax[2] = {-FLT_MAX, -FLT_MAX};

    float clamp_max[2] = {float(width-1), float(height-1)};
    float clamp_min[2] = {0, 0};

    size_t real_width = width;
    if(roi.width > 0 && roi.height > 0){  // depth will be flipped
        clamp_min[0] = width - (roi.x + roi.width) + 1; // +1 avoid out of roi
        clamp_min[1] = height - (roi.y + roi.height) + 1;
        clamp_max[0] = width - roi.x;
        clamp_max[1] = height - roi.y;
        real_width = roi.width;
    }


    for (int i=0; i<3; i++) {
        for (int j=0; j<2; j++) {
            bboxmin[j] = std__max(clamp_min[j], std__min(bboxmin[j], pts2[i][j]));
            bboxmax[j] = std__min(clamp_max[j], std__max(bboxmax[j], pts2[i][j]));
        }
    }

    size_t P[2];
    for(P[1] = size_t(bboxmin[1]+0.5f); P[1]<=bboxmax[1]; P[1] += 1){
        for(P[0] = size_t(bboxmin[0]+0.5f); P[0]<=bboxmax[0]; P[0] += 1){
            Model::float3 bc_screen  = barycentric(pts2[0], pts2[1], pts2[2], P);

            if (bc_screen.x<-0.0f || bc_screen.y<-0.0f || bc_screen.z<-0.0f ||
                    bc_screen.x>1.0f || bc_screen.y>1.0f || bc_screen.z>1.0f ) continue;

            Model::float3 bc_over_z = {bc_screen.x/last_row.x, bc_screen.y/last_row.y, bc_screen.z/last_row.z};

            // refer to https://en.wikibooks.org/wiki/Cg_Programming/Rasterization, Perspectively Correct Interpolation
            float frag_depth = -(dev_tri.v0.z*bc_over_z.x + dev_tri.v1.z*bc_over_z.y + dev_tri.v2.z*bc_over_z.z)
                    /(bc_over_z.x + bc_over_z.y + bc_over_z.z);

            size_t x_to_write = (width - P[0] - roi.x);
            size_t y_to_write = (height - P[1] - roi.y);

            int32_t depth = int32_t(frag_depth/**1000*/ + 0.5f);
            int32_t& depth_to_write = depth_entry[x_to_write+y_to_write*real_width];

            atomicMin(&depth_to_write, depth);
        }
    }
}

__global__ void render_triangle(Model::Triangle* device_tris_ptr, size_t device_tris_size,
                                Model::mat4x4* device_poses_ptr, size_t device_poses_size,
                                int32_t* depth_image_vec, size_t width, size_t height, const Model::mat4x4 proj_mat,
                                 const Model::ROI roi){
    size_t pose_i = blockIdx.y;
    size_t tri_i = blockIdx.x*blockDim.x + threadIdx.x;

    if(tri_i>=device_tris_size) return;
//    if(pose_i>=device_poses_size) return;

    size_t real_width = width;
    size_t real_height = height;
    if(roi.width > 0 && roi.height > 0){
        real_width = roi.width;
        real_height = roi.height;
    }

    int32_t* depth_entry = depth_image_vec + pose_i*real_width*real_height; //length: width*height 32bits int
    Model::mat4x4* pose_entry = device_poses_ptr + pose_i; // length: 16 32bits float
    Model::Triangle* tri_entry = device_tris_ptr + tri_i; // length: 9 32bits float

    // model transform
    Model::Triangle local_tri = transform_triangle(*tri_entry, *pose_entry);
//    if(normal_functor::is_back(local_tri)) return; //back face culling, need to be disable for not well defined surfaces?

    // assume last column of projection matrix is  0 0 -1 0
    Model::float3 last_row = {
        -local_tri.v0.z,
        -local_tri.v1.z,
        -local_tri.v2.z
    };
    // projection transform
    local_tri = transform_triangle(local_tri, proj_mat);

    rasterization(local_tri, last_row, depth_entry, width, height, roi);
}

std::vector<int32_t> render_cuda(const std::vector<Model::Triangle>& tris,const std::vector<Model::mat4x4>& poses,
                            size_t width, size_t height, const Model::mat4x4& proj_mat, const Model::ROI roi){

    const size_t threadsPerBlock = 256;

    thrust::device_vector<Model::Triangle> device_tris = tris;
    thrust::device_vector<Model::mat4x4> device_poses = poses;

    size_t real_width = width;
    size_t real_height = height;
    if(roi.width > 0 && roi.height > 0){
        real_width = roi.width;
        real_height = roi.height;
        assert(roi.x + roi.width <= width && "roi out of image");
        assert(roi.y + roi.height <= height && "roi out of image");
    }
    // atomic min only support int32
    thrust::device_vector<int32_t> device_depth_int(poses.size()*real_width*real_height, INT_MAX);
    {
        Model::Triangle* device_tris_ptr = thrust::raw_pointer_cast(device_tris.data());
        Model::mat4x4* device_poses_ptr = thrust::raw_pointer_cast(device_poses.data());
        int32_t* depth_image_vec = thrust::raw_pointer_cast(device_depth_int.data());

        dim3 numBlocks((tris.size() + threadsPerBlock - 1) / threadsPerBlock, poses.size());
        render_triangle<<<numBlocks, threadsPerBlock>>>(device_tris_ptr, tris.size(),
                                                        device_poses_ptr, poses.size(),
                                                        depth_image_vec, width, height, proj_mat, roi);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    std::vector<int32_t> result_depth(poses.size()*real_width*real_height);
    {
        thrust::transform(device_depth_int.begin(), device_depth_int.end(),
                          device_depth_int.begin(), max2zero_functor());
        thrust::copy(device_depth_int.begin(), device_depth_int.end(), result_depth.begin());
    }

    return result_depth;
}

thrust::device_vector<int32_t> render_cuda_keep_in_gpu(const std::vector<Model::Triangle>& tris,const std::vector<Model::mat4x4>& poses,
                            size_t width, size_t height, const Model::mat4x4& proj_mat, const Model::ROI roi){

    const size_t threadsPerBlock = 256;

    thrust::device_vector<Model::Triangle> device_tris = tris;
    thrust::device_vector<Model::mat4x4> device_poses = poses;

    size_t real_width = width;
    size_t real_height = height;
    if(roi.width > 0 && roi.height > 0){
        real_width = roi.width;
        real_height = roi.height;
    }
    // atomic min only support int32
    thrust::device_vector<int32_t> device_depth_int(poses.size()*real_width*real_height, INT_MAX);
    {
        Model::Triangle* device_tris_ptr = thrust::raw_pointer_cast(device_tris.data());
        Model::mat4x4* device_poses_ptr = thrust::raw_pointer_cast(device_poses.data());
        int32_t* depth_image_vec = thrust::raw_pointer_cast(device_depth_int.data());

        dim3 numBlocks((tris.size() + threadsPerBlock - 1) / threadsPerBlock, poses.size());
        render_triangle<<<numBlocks, threadsPerBlock>>>(device_tris_ptr, tris.size(),
                                                        device_poses_ptr, poses.size(),
                                                        depth_image_vec, width, height, proj_mat, roi);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    thrust::transform(device_depth_int.begin(), device_depth_int.end(),
                      device_depth_int.begin(), max2zero_functor());

    return device_depth_int;
}

}

